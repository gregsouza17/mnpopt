#include <stdio.h>
#include <stdbool.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>



extern __shared__ double cache[];

__global__ void kernel(int *Ss, int *Nn, int *mask, double *xyz,
		       double *cost){

  //block idx in 0, S
  //thread idx X 0,S and thread Idx Y 0,N
  //mask (N*k + j), cost( i1) xyz (N*S*k + S*j + i2)
  //cache =[threadIdx.x + threadIdx.y*blockDim.x]

  double temp=0;
  long int i1,i2,j, joffset=blockDim.y,i2offset=blockDim.x;
  int k, N=*Nn, S = *Ss, cacheIndex, cIndexMax;
  i1 = blockIdx.x; i2 = threadIdx.x;

  cacheIndex = threadIdx.x + threadIdx.y*blockDim.x;
  cIndexMax = blockDim.x*blockDim.y;


  while(i2<S){
   
    j = threadIdx.y;
    while(j<N){
      if (i1!=i2){ 
	// temp=0; 
	for(k=0; k<3; k++){
      
	  if( mask[k*N+j] ){ 
	
	//	temp+=1; 

	    //  printf("%f \t", xyz[i1+S*(j+N*k)]);
	    temp+=
	      (xyz[i1+S*(j+N*k)] - xyz[i2+S*(j+N*k)])* 
	      (xyz[i1+S*(j+N*k)] - xyz[i2+S*(j+N*k)]) ; 
	
	  } //if mask 
	  __syncthreads();
	} //k

      } //if i1!=i2
  
   
      __syncthreads();
      j+=joffset;
  } //while j<N;
    __syncthreads();
   i2+=i2offset;
  } //while i2<S

  __syncthreads();
   cache[cacheIndex]+= temp;


 

 
  //Somar todos os indices do cache aqui
  //ofset separado

   __syncthreads();
  int i = cIndexMax/2;
  while (i != 0) {
    if (cacheIndex < i)
      cache[cacheIndex] += cache[cacheIndex + i];
    __syncthreads();
    i /= 2;
    
  }


  __syncthreads();

  cost[i1] = cache[0];
 
}




extern "C"
void cslwcost_(int *N_atoms, int *Size_trj, int mask[],
	       double xyz[], double cost[], double *ctime)
{
  /*  A c module to be called from the
    subroutine Most_Representative_Configuration
    in stat.f (or statslw.f).

    Input: Adresses of Number of Atoms (N_Atoms),
    Size of Trajectorie Size_trj
    Boolean Mask (3, Numbe of atoms)
    Positions xyz (3, Number of atoms, Size of trj)
    Cost array

    Output: Sums the costs of each position in xyz(:,:, i1) in the cost(i1)*/
  
  int *dev_S, *dev_N;

  hipMalloc((void **)&dev_S, sizeof(int));
  hipMalloc((void **)&dev_N, sizeof(int));

  hipMemcpy(dev_S, Size_trj, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_N, N_atoms, sizeof(int), hipMemcpyHostToDevice);

  int *dev_mask;

  hipMalloc((void **)&dev_mask, 3*(*N_atoms)*sizeof(int));
  
  hipMemcpy(dev_mask,  mask,  3*(*N_atoms)*sizeof(int),
	     hipMemcpyHostToDevice);

  double *dev_xyz, *dev_cost;

  hipMalloc((void **)&dev_xyz, 3*(*N_atoms)*(*Size_trj)*sizeof(double));

  hipMalloc((void **)&dev_cost, *Size_trj*sizeof(double));

  hipMemcpy(dev_xyz, xyz, 3*(*N_atoms)*(*Size_trj)*sizeof(double),
	     hipMemcpyHostToDevice);

  int trdX = 2, trdY=2, cacheSize;
  cacheSize = trdX*trdY;

  dim3 threads(trdX, trdY);

  kernel<<<*Size_trj, threads, cacheSize*sizeof(double)>>>
    (dev_S, dev_N, dev_mask, dev_xyz, dev_cost);

  hipMemcpy(cost, dev_cost, *Size_trj*sizeof(double),
	     hipMemcpyDeviceToHost);
  

    FILE *fp; 
   fp = fopen("tst/GPU_RMSDCPU.dat", "w+"); 
   int i1;
   for(i1=0; i1<*Size_trj; i1++) 
     fprintf(fp, "%d %f \n", i1, cost[i1]); 
  
  hipFree(dev_S); hipFree(dev_N); hipFree(dev_xyz);
  hipFree(dev_cost); hipFree(dev_mask);

  

  



}
  
  /*
  FILE *fpCost;
  fpCost = fopen("tst/Ccostcpu.dat", "w+");
  for(i1 = 0;   i1<*Size_trj;    ++i1)
  fprintf(fpCost, "%d \t %f \n",     i1,  cost[i1] ); */



  

 





//TESTS ===========================
  
  /* FILE *fp; */
  /* fp = fopen("tst/maksCtst.dat", "w+"); */

  //Test for xyz being passed corretclty from fortran to the C moduel
  
  /* for (i1 = 0; i1 < *Size_trj; i1+=103) { */
  /*   for(j =0; j< *N_atoms; j+=101){ */
  /*     for(k=0; k<3; k++){ */
  /* 	fprintf(fp, "%d %d: %f" , i1,j, xyz[k][j][i1]); */
  /*     } */
  /*     fprintf(fp, "\n"); */
  /*   } */
  /*   fprintf(fp, "\n"); */
  /* } */



  /* for (i1 = 0; i1 < *Size_trj; ++i1) { */
  /*   fprintf(fp, "%d : %f\n",i1, cost[i1]); */
  /* } */


 //Cost computing
  /* soma = 0; */

  /* for (i1 = 0  ; i1 < *Size_trj  ; ++i1) { */
  /*   for (i2 = 0;   i2< *Size_trj  ; ++i2) { */
  /*     if(i1 != i2){ */
  /* 	soma = 0; */
  /* 	for(j=0;   j<*N_atoms;   j++){ */
  /* 	  for(k=0;   k<3;    k++){ */

  /* 	     if(  mask [k*N+j] ){ */
  /* 	       soma+=(xyz[i1+ S*(j + N*k)] -xyz[i2+ S*(j + N*k)])* */
  /* 		 (xyz[i1+ S*(j + N*k)] -xyz[i2+ S*(j + N*k)]) ; */
  /* 	     } */
	    
  /* 	  } //for k */
  /* 	} //for j */
  /*      	cost[i1]+=soma; */
  /*     } //for if */
  /*   } //for i2 */
  /* } //for i1 */


/*   t = clock() - t; */
/*   *ctime = (double) t/CLOCKS_PER_SEC; */
